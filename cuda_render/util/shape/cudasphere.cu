#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "cudashape.cu.h"
using namespace optix;
rtDeclareVariable(float, radius, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

__device__ __inline__ bool Quadratic(float A, float B, float C, float *t0, float *t1) {
    // Find quadratic discriminant
    float discrim = B * B - 4.f * A * C;
    if (discrim < 0.) return false;
    float rootDiscrim = sqrtf(discrim);

    // Compute quadratic _t_ values
    float q;
    if (B < 0) q = -.5f * (B - rootDiscrim);
    else       q = -.5f * (B + rootDiscrim);
    *t0 = q / A;
    *t1 = C / q;
    if (*t0 > *t1){
        float temp=*t0;
        *t0=*t1;
        *t1=temp;
    }
    return true;
}

RT_PROGRAM void sphere_intersect(int primIdx)
{
    float A=dot(ray.direction, ray.direction);
    float B=2.f*dot(ray.direction, ray.origin);
    float C=dot(ray.origin, ray.origin)-radius*radius;
    float t0,t1;
    if (!Quadratic(A, B, C, &t0, &t1))
        return;

    bool check_second=true;
    
    if(rtPotentialIntersection(t0)){
        float3 phit=ray.origin+t0*ray.direction;
        if (phit.x == 0.f && phit.y == 0.f) phit.x = 1e-5f * radius;
        float phi = atan2f(phit.y, phit.x);
        if (phi < 0.) phi += 2.f*M_PI;
        float u=phi/(2.0f*M_PI);
        float theta = acosf(clamp(phit.z / radius, -1.f, 1.f));
        float v = (theta)/(M_PI);
        uv=make_float2(u, v);

        shading_normal = geometry_normal = phit/radius;
        dpdu=make_float3(-shading_normal.y, shading_normal.x, 0.f);
        dpdv=cross(shading_normal, dpdu);
        if(rtReportIntersection(0)){
            check_second=false;
        }
    }
    if(check_second) {
        if(rtPotentialIntersection(t1)){
            float3 phit=ray.origin+t1*ray.direction;
            if (phit.x == 0.f && phit.y == 0.f) phit.x = 1e-5f * radius;
            float phi = atan2f(phit.y, phit.x);
            if (phi < 0.) phi += 2.f*M_PI;
            float u=phi/(2.0f*M_PI);
            float theta = acosf(clamp(phit.z / radius, -1.f, 1.f));
            float v = (theta)/(M_PI);
            uv=make_float2(u, v);

            shading_normal = geometry_normal = phit/radius;
            dpdu=make_float3(-shading_normal.y, shading_normal.x, 0.f);
            dpdv=cross(shading_normal, dpdu);
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void sphere_bounds (int, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->m_min = make_float3(-radius);
  aabb->m_max = make_float3(radius);
}
