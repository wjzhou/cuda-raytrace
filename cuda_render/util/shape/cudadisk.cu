#include "hip/hip_runtime.h"
#include <optix_world.h>
using namespace optix;
#define INV_TWOPI  0.15915494309189533577f

//input
rtDeclareVariable(float, height, attribute height, );
rtDeclareVariable(float, radius, attribute radius, );
rtDeclareVariable(float, innerRadius, attribute innerRadius, );
rtDeclareVariable(float, phiMax, attribute phiMax, );

//output
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometry_normal, attribute geometry_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, uv, attribute uv, );//tex coordinate
//rtDeclareVariable(float3, p, attribute p, ); //point position
rtDeclareVariable(float3, dpdu, attribute dpdu, );
rtDeclareVariable(float3, dpdv, attribute dpdv, );

RT_PROGRAM void disk_intersect(int primIdx)
{
    //optix::Ray ray; // to fool the vx assist..
    // Already in object space
    // Compute plane intersection for disk
    if (fabsf(ray.direction.z) < 1e-7) return;
    float thit = (height - ray.origin.z) / ray.direction.z;

    if(rtPotentialIntersection(thit)){
    // See if hit point is inside disk radii and $\phimax$
        float3 phit = ray.origin + thit*ray.direction;
        float dist2 = phit.x * phit.x + phit.y * phit.y;
        if (dist2 > radius * radius || dist2 < innerRadius * innerRadius)
            return;

    // Test disk $\phi$ value against $\phimax$
        float phi = atan2f(phit.y, phit.x);
        if (phi < 0) phi += 2.f * M_PI;
          if (phi > phiMax) return;

    // Find parametric representation of disk hit
        float oneMinusV = ((sqrtf(dist2)-innerRadius) /
                       (radius-innerRadius));
        float invOneMinusV = (oneMinusV > 0.f) ? (1.f / oneMinusV) : 0.f;
        uv=make_float2(phi/phiMax, 1.f-oneMinusV);
        dpdu=make_float3(-phiMax * phit.y, phiMax * phit.x, 0.f);
        dpdu *= phiMax * INV_TWOPI;
        rtTransformVector(RT_OBJECT_TO_WORLD, dpdu);
        dpdv=make_float3(-phit.x * invOneMinusV, -phit.y * invOneMinusV, 0.f);
        dpdv *= (radius - innerRadius) / radius;
        rtTransformVector(RT_OBJECT_TO_WORLD, dpdv);

        //I do not use the rayEpsilon currently, may use it in future
        //*rayEpsilon = 5e-4f * *tHit;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void disk_intersectP(int primIdx)
{
    if (fabsf(ray.direction.z) < 1e-7) return;
    float thit = (height - ray.origin.z) / ray.direction.z;
    if (rtPotentialIntersection(thit)){

    // See if hit point is inside disk radii and $\phimax$
    float3 phit = ray.origin + thit*ray.direction;
    float dist2 = phit.x * phit.x + phit.y * phit.y;
    if (dist2 > radius * radius || dist2 < innerRadius * innerRadius)
        return;

    // Test disk $\phi$ value against $\phimax$
    float phi = atan2f(phit.y, phit.x);
    if (phi < 0) phi += 2. * M_PI;
    if (phi > phiMax) return;
    rtReportIntersection(0);
    }
}


RT_PROGRAM void disk_bounds (int primIdx, float result[6])
{
  optix::Aabb* aabb = (optix::Aabb*)result;
 
    aabb->m_min = make_float3(-radius, -radius, height);
    aabb->m_max = make_float3( radius,  radius, height);
 
}

struct SampleResult{
    float3 point;
    float3 normal;
} ;
/*RT_CALLABLE_PROGRAM SampleResult disk_sample(float u1, float u2 )
{
    Point p;
    ConcentricSampleDisk(u1, u2, &p.x, &p.y);
    p.x *= radius;
    p.y *= radius;
    p.z = height;
    *Ns = Normalize((*ObjectToWorld)(Normal(0,0,1)));
    if (ReverseOrientation) *Ns *= -1.f;
    return (*ObjectToWorld)(p);
    SampleResult result;
    return result;

}*/
