#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "cudashape.cu.h"
using namespace optix;


//input
rtDeclareVariable(float2, invRadius2, , );
rtDeclareVariable(float, innerRadius, , );
rtDeclareVariable(float, phiMax, , );
rtDeclareVariable(float3, worldx, , );
rtDeclareVariable(float3, worldy, , );
rtDeclareVariable(float3, worldz, , );
rtDeclareVariable(float3, worldo, , );
rtDeclareVariable(float, moffset, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void disk_intersect(int primIdx)
{
    //divid 0 is ok, the IEEE 754 will get inf, and the if will fail
    float thit=(moffset-dot(worldz,ray.origin))/dot(worldz,ray.direction);
    if (thit>ray.tmin && thit<ray.tmax){
        float3 phit = ray.origin + thit*ray.direction;

        float3 localPhit=phit-worldo;
        float localx=dot(localPhit, worldx)*invRadius2.x;
        float localy=dot(localPhit, worldy)*invRadius2.y;

        float dist2=localx*localx+localy*localy;
        if (dist2 > 1.f || dist2 < innerRadius*innerRadius)
        return;

        float phi = atan2f(localy, localx);
        if (phi < 0) phi += 2.f * M_PI;
        if (phi > phiMax) return;
        if(rtPotentialIntersection(thit)){
           
            aShadingNormal=aGeometryNormal=worldz;
            float oneMinusV = ((sqrt(dist2)-innerRadius) /
                (1.f-innerRadius));
            //float invOneMinusV = (1.f / oneMinusV);
             aUv=make_float2(phi/phiMax, 1.f-oneMinusV);
            aDpdu = -localy*worldx+localx*worldy;
            //dpdu *= phiMax * INV_TWOPI;
            aDpdv = -localx*worldx-localy*worldy;
            //dpdv *= (radius - innerRadius) / radius;*/
            rtReportIntersection(0);
        }
    }    
}

/*    //optix::Ray ray; // to fool the vx assist..
// Already in object space
// Compute plane intersection for disk
if (fabsf(ray.direction.z) < 1e-7) return;
float thit = (height - ray.origin.z) / ray.direction.z;

if(rtPotentialIntersection(thit)){
// See if hit point is inside disk radii and $\phimax$
float3 phit = ray.origin + thit*ray.direction;
float dist2 = phit.x * phit.x + phit.y * phit.y;
if (dist2 > radius * radius || dist2 < innerRadius * innerRadius)
return;

// Test disk $\phi$ value against $\phimax$
float phi = atan2f(phit.y, phit.x);
if (phi < 0) phi += 2.f * M_PI;
if (phi > phiMax) return;

// Find parametric representation of disk hit
float oneMinusV = ((sqrtf(dist2)-innerRadius) /
(radius-innerRadius));
float invOneMinusV = (oneMinusV > 0.f) ? (1.f / oneMinusV) : 0.f;
uv=make_float2(phi/phiMax, 1.f-oneMinusV);
dpdu=make_float3(-phiMax * phit.y, phiMax * phit.x, 0.f);
dpdu *= phiMax * INV_TWOPI;
rtTransformVector(RT_OBJECT_TO_WORLD, dpdu);
dpdv=make_float3(-phit.x * invOneMinusV, -phit.y * invOneMinusV, 0.f);
dpdv *= (radius - innerRadius) / radius;
rtTransformVector(RT_OBJECT_TO_WORLD, dpdv);

//I do not use the rayEpsilon currently, may use it in future
//*rayEpsilon = 5e-4f * *tHit;
rtReportIntersection(0);
}*/

RT_PROGRAM void disk_bounds (int primIdx, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    float3 p0=worldo+worldx+worldy;
    float3 p1=worldo+worldx-worldy;
    float3 p2=worldo-worldx+worldy;
    float3 p3=worldo-worldx-worldy;
    aabb->m_min = fminf(fminf(fminf(p0, p1), p2),p3);
    aabb->m_max = fmaxf(fmaxf(fmaxf(p0, p1), p2),p3);
}

