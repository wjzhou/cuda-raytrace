#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "cudashape.cu.h"
using namespace optix;

rtBuffer<float3> bVertices;
rtBuffer<float3> bNormals;
rtBuffer<float2> bUvs;
rtBuffer<int3>   bIndices;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void trianglemesh_intersect( int primIdx )
{
    int3 idx = bIndices[primIdx];
    float3 p0 = bVertices[idx.x];
    float3 p1 = bVertices[idx.y];
    float3 p2 = bVertices[idx.z];

    float3 n;
    float  t, beta, gamma;
    //use the ray-triangle intersection method from the optix library.
    //The comments of that function memtioned branch-less, thus, it
    //should be more efficient.
    if(intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)){
        if(rtPotentialIntersection(t)) {
            float2 uv0, uv1, uv2;
            if (bUvs.size() == 0) {
                //assume t0=(0,0), t1=(1,0), t2=(0,1)
                //texcoord = make_float2(beta+gamma);
                uv0=make_float2(0.0f, 0.0f);
                uv1=make_float2(1.0f, 0.0f);
                uv2=make_float2(0.0f, 1.0f);
            } else {
                uv0=bUvs[idx.x];
                uv1=bUvs[idx.y];
                uv2=bUvs[idx.z];
                //texcoord = t1*beta + t2*gamma + t0*(1.0f-beta-gamma) ;
            }

            float du1=uv0.x-uv2.x;
            float du2=uv1.x-uv2.x;
            float dv1=uv0.y-uv2.y;
            float dv2=uv1.y-uv2.y;

            float3 dp1=p0-p2;
            float3 dp2=p1-p2;
            float determinant=du1*dv2-dv1*du2;
            //degenerate triangle, choose the dpdu arbitary
            if (determinant == 0.0f){
                if (fabsf(n.x) > fabsf(n.y)) {
                    float invLen = 1.f/sqrtf(n.x*n.x+n.z*n.z);
                    dpdu = make_float3(-n.z*invLen, 0.f, n.x*invLen);
                }
                else {
                    float invLen = 1.f/sqrtf(n.y*n.y+n.z*n.z);
                    dpdu = make_float3(0.f, n.z*invLen, n.y*invLen);
                }
                dpdv =normalize(cross(n, dpdu));
            }else{
                float invdet = 1.f / determinant;
                dpdu = ( dv2 * dp1 - dv1 * dp2) * invdet;
                dpdv = (-du2 * dp1 + du1 * dp2) * invdet;

            }
             uv=(1-beta-gamma)*uv0+beta*uv1+gamma*uv2;
             //todo alphaTexture
             geometry_normal=normalize(n);
             if (bNormals.size() == 0) {
                 shading_normal = geometry_normal;
             } else {
                 float3 n0 = bNormals[idx.x];
                 float3 n1 = bNormals[idx.y];
                 float3 n2 = bNormals[idx.z];
                 shading_normal = normalize(n1*beta + n2*gamma + n0*(1.0f-beta-gamma));
                 //shading override
             }
             //p=ray.origin+t*ray.direction;
             rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void trianglemesh_bounds (int primIdx, float result[6])
{
  const int3 idx = bIndices[primIdx];

  const float3 p0 = bVertices[idx.x];
  const float3 p1 = bVertices[idx.y];
  const float3 p2 = bVertices[idx.z];
  const float area = length(cross(p1-p0, p2-p0));

  optix::Aabb* aabb = (optix::Aabb*)result;
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf(fminf(p0, p1), p2);
    aabb->m_max = fmaxf(fmaxf(p0, p1), p2);
  } else {
    aabb->invalidate();
  }
}

