#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "camera.cu.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);
rtBuffer<CudaRayDifferential, 2> bRays;
RT_CALLABLE_PROGRAM CudaRayDifferential cameraRay()
{
    return bRays[launchIndex];
}

// Stubs only needed for sm_1x
#if __CUDA_ARCH__ < 200
__global__ void checker_color_stub()
{
  cameraRay();
}
#endif