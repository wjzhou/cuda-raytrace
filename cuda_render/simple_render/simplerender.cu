#include "hip/hip_runtime.h"
#include "util/camera/camera.cu.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

struct RtRayPayload
{
     optix::uint ray_depth;
};

rtBuffer<float3, 2>  bOutput;
rtDeclareVariable(rtObject, top_group, ,);
rtDeclareVariable(float, scene_epsilon, ,);
rtCallableProgram(CudaRayDifferential, cameraRay, ());
RT_PROGRAM void simple_camera(){
    CudaRayDifferential rd=cameraRay();
    optix::Ray ray(rd.o, rd.d, 0, scene_epsilon);
    RtRayPayload pld;
    pld.ray_depth=0u;
    rtTrace(top_group, ray, pld);
}

__device__ __inline__ float3 WorldtoLocal(const float3& v,
    const float3& nn, const float3& sn, const float3& tn)
{
    return make_float3(dot(v, sn), dot(v,tn), dot(v,nn));
}


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometry_normal, attribute geometry_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float2, uv, attribute uv, );//tex coordinate
rtDeclareVariable(float3, dpdu, attribute dpdu, );
rtDeclareVariable(float3, dpdv, attribute dpdv, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);

RT_PROGRAM void simple_cloest_hit()
{

    float3 world_shading_normal = normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometry_normal = normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, geometry_normal));
    //float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometry_normal
    float costha=dot(world_shading_normal, ray.direction);
    if(costha>0.0f)
        bOutput[launchIndex]=make_float3(costha,0.0f,0.0f);
    else
        bOutput[launchIndex]=make_float3(0.2f,0.2f,0.2f);

    //bOutput[launchIndex]=t*ray.direction+ray.origin;


    //bOutput[launchIndex]=make_float3(1.0f, 0.0f, 0.0f);

}

RT_PROGRAM void simple_miss()
{
    bOutput[launchIndex]=make_float3(0.3f, 0.3f, 0.0f);
}


RT_PROGRAM void simple_exception()
{
  bOutput[launchIndex] = make_float3(1.0f, 1.0f, 0.0f);
  rtPrintExceptionDetails();
}


