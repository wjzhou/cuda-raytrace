#include "hip/hip_runtime.h"
#include "util/camera/camera.cu.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "util/light/cudalight.cu.h"
#include "util/camera/camera.cu.h"
#include "util/material/cudamaterial.cu.h"
#include "util/shape/cudashape.cu.h"
using namespace optix;

struct RtRayPayload
{
     optix::uint ray_depth;
};

rtBuffer<float3, 2>  bOutput;
rtDeclareVariable(rtObject, top_group, ,);
rtDeclareVariable(float, scene_epsilon, ,);
//rtCallableProgram(CudaRayDifferential, cameraRay, ());
//rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);
rtBuffer<CudaRayDifferential, 2> bRays;

RT_PROGRAM void simple_camera(){
    CudaRayDifferential rd=bRays[launchIndex];
    optix::Ray ray(rd.o, rd.d, 0, scene_epsilon);
    RtRayPayload pld;
    pld.ray_depth=0u;
    rtTrace(top_group, ray, pld);
}

__device__ __inline__ float3 WorldtoLocal(const float3& v,
    const float3& nn, const float3& sn, const float3& tn)
{
    return make_float3(dot(v, sn), dot(v,tn), dot(v,nn));
}

struct ShadowPRD
{
  float attenuation;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );

//rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);

RT_PROGRAM void simple_cloest_hit()
{
    CudaSpectrum L=black();
    L.x=L.y=L.z=0.0f;
    const float3 point=ray.origin+ray.direction*t;

    float3 world_shading_normal = normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometry_normal = normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, geometry_normal));
    
    int totalLight=lightSize();
    //rtPrintf("1 %f,%f,%f\n", L.x,L.y,L.z);
    for (int i=0; i<totalLight; ++i)
    {
        float3 uwi;
        float pdf;
        CudaSpectrum li=Sample_L(i, point, uwi, pdf);
        Ray shadowRay(point, uwi, 1, 0.001f, 1.0f-0.001f);
        ShadowPRD pld;
        pld.attenuation=1.0f;
        rtTrace(top_group, shadowRay, pld);
        float3 wi=normalize(uwi);
        float3 wo=normalize(-ray.direction);
        //if(pld.attenuation>0.f)
        L+=pld.attenuation*fabs(dot(world_shading_normal, wi))*f(wo, wi)*li;
        //rtPrintf("2 %f,%f,%f,%f\n", L.x,L.y,L.z, li.x);
    }

    bOutput[launchIndex]=L;
#ifdef DEBUG_KERNEL
    if(isPrint()){
        rtPrintf("\nDL:%f %f %f", L.x, L.y, L.z);
    }
#endif
}

RT_PROGRAM void simple_miss()
{
//bOutput[launchIndex]=make_float3(0.3f, 0.3f, 0.0f);
    bOutput[launchIndex]=make_float3(0.0f);
}



RT_PROGRAM void simple_shadow_any_hit()
{
    shadow_prd.attenuation=0.f;
    rtTerminateRay();
}


RT_PROGRAM void simple_exception()
{
  bOutput[launchIndex] = make_float3(1.0f, 1.0f, 0.0f);
  rtPrintExceptionDetails();
}


