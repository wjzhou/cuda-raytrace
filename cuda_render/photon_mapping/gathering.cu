#include "hip/hip_runtime.h"
#include "optix_world.h"
#include "photonmapping.h"
#include "util/material/cudamaterial.cu.h"
#include "util/util.cu.h"
#include "util/light/cudalight.cu.h"
rtBuffer<RayTracingRecord, 2> bRayTracingOutput;
rtBuffer<float3, 2>  bOutput;
rtDeclareVariable(CudaPhoton*, indirectPhotonmap, , );
#define MAX_DEPTH 20

__device__ __inline__ float kernel(float dist2, float maxDist2)
{
     float s = (1.f - dist2 / maxDist2);
     return 3.f * INV_PI * s * s;
}

__device__ __inline__ CudaSpectrum processPhoton(const RayTracingRecord& record, const CudaPhoton& photon, float dist2, float maxDist2)
{
    float k = kernel(dist2, maxDist2);
    if(isPrint()){
        rtPrintf("\nalpha:%f, %f, %f",photon.alpha.x, photon.alpha.y, photon.alpha.z);
    }
    return k/(maxDist2)*fabs(dot(record.shadingNormal, photon.wi))*f(record.material, record.materialParameter, -record.direction, photon.wi)*photon.alpha;
}

__device__ __inline__ CudaSpectrum KdTreeLookup(RayTracingRecord& rec, int nLookup, float maxDist2)
{
    unsigned int stack[MAX_DEPTH];
    unsigned int stack_current = 0;
    unsigned int nodeNum = 0;
    CudaSpectrum L=black();
    //float3& p=rec.position;
    float3 p=make_float3(0.f,0.f,0.f);
#define push_node(N) stack[stack_current++] = (N)
#define pop_node()   stack[--stack_current]
    push_node( 0 );
    do {
        if(nodeNum>(1<<12)){
            nodeNum=pop_node();
            continue;
        }
        CudaPhoton* node=&indirectPhotonmap[nodeNum];
        int axis=node->splitAxis;

        float dist2=DistanceSquared(node->p, p);
        if(dist2 < maxDist2){
              L+=processPhoton(rec, *node, dist2, maxDist2);
        }
        

        if(axis<3){
            /*float p_axis, nodeP_axis;
            switch (axis)
            {
            case 0:
                p_axis=p.x;
                nodeP_axis=node->p.x;
                break;
            case 1:
                p_axis=p.y;
                nodeP_axis=node->p.y;
                break;
            case 2:
                p_axis=p.z;
                nodeP_axis=node->p.z;
                break;
            }*/
            float p_axis=reinterpret_cast<float*>(&p)[axis];
            float nodeP_axis=reinterpret_cast<float*>(&node->p)[axis];
            float dist2 = (p_axis - nodeP_axis)*(p_axis - nodeP_axis);
            
            if (p_axis <= nodeP_axis){
                if (dist2 < maxDist2 && node->rightChild < PHOTON_MAX_RIGHT_CHILD){
                    push_node(node->rightChild); 
                }
                if (node->hasLeftChild){
                    nodeNum=nodeNum+1;
                }else{
                    nodeNum=pop_node();
                }

            } else {
                if (dist2 < maxDist2 && node->hasLeftChild){
                    push_node(nodeNum+1);
                }
                if (node->rightChild < PHOTON_MAX_RIGHT_CHILD){
                    nodeNum=node->rightChild;
                }else{
                    nodeNum=pop_node();
                }
            }
        }
        else{
            nodeNum=pop_node();
        }

    } while (nodeNum);
    return L;
#undef push_node
#undef pop_node
}

__device__ __inline__ CudaSpectrum LPhoton(RayTracingRecord& rec, int nLookup, float maxDist2)
{
    return KdTreeLookup(rec, nLookup, maxDist2);
    //return black();
}

struct ShadowPRD
{
    float attenuation;
};
rtDeclareVariable(rtObject, top_group, ,);
rtDeclareVariable(float, scene_epsilon, ,);
__device__ __inline__ CudaSpectrum directLight(const RayTracingRecord& rec){
    CudaSpectrum L=black();
    const float3& point=rec.position;

    float3 world_shading_normal = rec.geometryNormal;
    float3 world_geometry_normal = rec.shadingNormal;

    int totalLight=lightSize();
    //rtPrintf("1 %f,%f,%f\n", L.x,L.y,L.z);
    for (int i=0; i<totalLight; ++i)
    {
        float3 uwi;
        float pdf;
        CudaSpectrum li=Sample_L(i, point, uwi, pdf);
        Ray shadowRay(point, uwi, PM_ShadowRayType, 0.001f, 1.0f-0.001f);
        ShadowPRD pld;
        pld.attenuation=1.0f;
        rtTrace(top_group, shadowRay, pld);
        float3 wi=normalize(uwi);
        float3 wo=normalize(-rec.direction);
        L+=pld.attenuation*fabs(dot(world_shading_normal, wi))*f(rec.material, rec.materialParameter, wo, wi)*li;
        //rtPrintf("2 %f,%f,%f,%f\n", L.x,L.y,L.z, li.x);
    }
    return L;
}

RT_PROGRAM void gathering_camera(){
    RayTracingRecord& rec=bRayTracingOutput[launchIndex];

    int rec_flags=rec.flags;
    // Check if this is hit point lies on an emitter or hit background 
    if( (rec_flags & RayTracingRecordFlageMISS) || (rec_flags & RayTracingRecordFlageException )) {
        bOutput[launchIndex] = make_float3(0.0f,0.0f,0.0f);
        return;
    }
    //rtPrintf("%ld", (long long)indirectPhotonmap);
    
    //bOutput[launchIndex]=directLight(rec)+LPhoton(rec, 8, 10.0f);
    CudaSpectrum DL=directLight(rec);
    CudaSpectrum IDL=LPhoton(rec, 8, 50.0f);
    if(isPrint()){
        rtPrintf("\nDL:%f %f %f, IDL: %f %f %f", DL.x, DL.y, DL.z, IDL.x, IDL.y, IDL.z);
    }
    //bOutput[launchIndex]=DL+IDL;
    bOutput[launchIndex]=DL+IDL;
}