#include "hip/hip_runtime.h"
#include "optix_world.h"
#include "photonmapping.h"
#include "util/material/cudamaterial.cu.h"
#include "util/util.cu.h"
#include "util/light/cudalight.cu.h"
rtBuffer<RayTracingRecord, 2> bRayTracingOutput;
rtBuffer<float3, 2>  bOutput;
rtDeclareVariable(CudaPhoton*, indirectPhotonmap, , );
#define MAX_DEPTH 40

__device__ __inline__ float kernel(float dist2, float maxDist2)
{
     float s = (1.f - dist2 / maxDist2);
     return 3.f * INV_PI * s * s;
}

__device__ __inline__ CudaSpectrum processPhoton(const RayTracingRecord& record, const CudaPhoton& photon, float dist2, float maxDist2)
{
    float k = kernel(dist2, maxDist2);
    rtPrintf("\nalpha:%f, %f, %f",photon.alpha.x, photon.alpha.y, photon.alpha.z);
    //return k/(maxDist2)*fabs(dot(record.shadingNormal, photon.wi))*f(record.material, record.materialParameter, -record.direction, photon.wi)*photon.alpha;
    return fabs(dot(record.shadingNormal, photon.wi))*f(record.material, record.materialParameter, -record.direction, photon.wi)*photon.alpha;
}

__device__ __inline__ CudaSpectrum KdTreeLookup(RayTracingRecord& rec, int& nLookup, float maxDist2)
{
    unsigned int stack[MAX_DEPTH];
    unsigned int stack_current = 0;
    unsigned int nodeNum = 0;
    CudaSpectrum L=black();
    float3& p=rec.position;
#define push_node(N) stack[stack_current++] = (N)
#define pop_node()   stack[--stack_current]
    push_node( 0 );
    do {
        CudaPhoton* node=&indirectPhotonmap[nodeNum];
        int axis=node->splitAxis;

        float dist2=DistanceSquared(node->p, p);
        if(dist2 < maxDist2){
            nLookup++;
            L+=processPhoton(rec, *node, dist2, maxDist2);
        }
        

        if(axis<3){
            /*float p_axis, nodeP_axis;
            switch (axis)
            {
            case 0:
                p_axis=p.x;
                nodeP_axis=node->p.x;
                break;
            case 1:
                p_axis=p.y;
                nodeP_axis=node->p.y;
                break;
            case 2:
                p_axis=p.z;
                nodeP_axis=node->p.z;
                break;
            }*/
            float p_axis=reinterpret_cast<float*>(&p)[axis];
            float nodeP_axis=reinterpret_cast<float*>(&node->p)[axis];
            float dist2 = (p_axis - nodeP_axis)*(p_axis - nodeP_axis);
            
            if (p_axis <= nodeP_axis){
                if (dist2 < maxDist2 && node->rightChild < PHOTON_MAX_RIGHT_CHILD){
                    push_node(node->rightChild); 
                }
                if (node->hasLeftChild){
                    nodeNum=nodeNum+1;
                }else{
                    nodeNum=pop_node();
                }

            } else {
                if (dist2 < maxDist2 && node->hasLeftChild){
                    push_node(nodeNum+1);
                }
                if (node->rightChild < PHOTON_MAX_RIGHT_CHILD){
                    nodeNum=node->rightChild;
                }else{
                    nodeNum=pop_node();
                }
            }
        }
        else{
            nodeNum=pop_node();
        }

    } while (nodeNum);
    return L;
#undef push_node
#undef pop_node
}

__device__ __inline__ CudaSpectrum LPhoton(RayTracingRecord& rec, int& nLookup, float maxDist2)
{
    return KdTreeLookup(rec, nLookup, maxDist2);
    //return black();
}

RT_PROGRAM void photonGatheringCamera(){
    RayTracingRecord& rec=bRayTracingOutput[launchIndex];

    int rec_flags=rec.flags;
    // Check if this is hit point lies on an emitter or hit background 
    if( (rec_flags & RayTracingRecordFlageMISS) || (rec_flags & RayTracingRecordFlageException )) {
        return;
    }
    
    int currentPhotons=0;
    CudaSpectrum IDL=LPhoton(rec, currentPhotons, rec.radius2);
    float alpha=0.7f;
    if(currentPhotons>0){
        int totalPhotons=rec.photon_count+alpha*currentPhotons;
        float ratio=totalPhotons/(rec.photon_count+currentPhotons);
        rec.radius2=rec.radius2*ratio;

        rec.flux=(rec.flux+IDL)*ratio;
        rec.photon_count=totalPhotons;
        rtPrintf("\nradius2:%f, currphotons:%d,  photons:%d IDL: %f %f %f", rec.radius2, currentPhotons, 
            rec.photon_count, rec.flux.x/rec.photon_count, rec.flux.y/rec.photon_count, rec.flux.z/rec.photon_count);
    }
}

rtDeclareVariable(float, emittingPhotons, ,);
RT_PROGRAM void finalGatheringCamera(){

    
    RayTracingRecord& rec=bRayTracingOutput[launchIndex];
    if( (rec.flags & RayTracingRecordFlageMISS) || (rec.flags & RayTracingRecordFlageException )) {
        bOutput[launchIndex] = make_float3(0.0f,0.0f,0.0f);
        return;
    }
    CudaSpectrum DL=rec.directLight;
    CudaSpectrum IDL=black();
    if(rec.photon_count!=0){
        IDL=rec.flux*INV_PI/(rec.radius2*emittingPhotons);
    }
        
    rtPrintf("\n------DL:%f %f %f,emittingPhotons:%f,  IDL: %f %f %f\n-----\n", 
        DL.x, DL.y, DL.z, emittingPhotons, IDL.x, IDL.y, IDL.z);
    bOutput[launchIndex]=DL+IDL;
}