#include "hip/hip_runtime.h"
#include "optix_world.h"

#include "util/light/cudalight.cu.h"
#include "util/util.cu.h"

#include "util/material/cudamaterial.cu.h"
#include "photonmapping.h"
using namespace optix;

rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_group, , );

#define dims 4
//PermutedHalton halton(6, rng);
/*
__device__ uint *haltonPermutation;
__device__ uint *b;
__device__ __inline__ float PermutedRadicalInverse(uint n, uint base, uint* p) 
{
    float val = 0;
    float invBase = 1.f / base, invBi = invBase;

    while (n > 0) {
        uint d_i = p[n % base];
        val += d_i * invBi;
        n *= invBase;
        invBi *= invBase;
    }
    return val;
}

static const float OneMinusEpsilon=0x1.fffffep-1;
__device__ __inline__ void haltonSample(uint n, float* out)
{
    uint *p = haltonPermutation;
    for (uint i = 0; i < dims; ++i) {
        out[i] = min(PermutedRadicalInverse(n, b[i], p), 
            OneMinusEpsilon);
        p += b[i];
    }
}*/

//use a standard halton sequence (a.k.a non-permuted) version at this moment
//this is determined and generate (1/2, 1/3, 1/5, 1/7) fot the (0,0) 
__device__ static const uint b[]={2,3,5,7,11,13}; 
__device__ __inline__ void haltonSample(uint n, float* out)
{
    for (uint i=0; i<dims; ++i){
        uint n1=n+1;
        float val = 0;
        uint base=b[i];
        float invBase = 1. / base, invBi = invBase;
        while (n1 > 0) {
            val += (n1%base) * invBi;
            n1 *= invBase;
            invBi *= invBase;
        }
        out[i]=val;
    }
}

rtDeclareVariable(uint, lightSourceIndex, , );
rtDeclareVariable(uint, photonTracinglaunchWidth, , );
rtDeclareVariable(uint, max_photon_count, ,);
rtDeclareVariable(uint, photonTracingEmittingPhotons, ,);

enum SampleIndex{LU1=0,LU2, U1,U2};

struct PhotonTraingPayLoad{
    CudaSpectrum alpha;
    bool specularPath;
    uint nIntersections;
    uint pm_index;
};

rtDeclareVariable(CudaPhoton*, indirectPhotonmap, , );
rtDeclareVariable(float*, photonTracingRandom, , );

RT_PROGRAM void photontracing_camera()
{
    uint  pm_index = (launchIndex.y * photonTracinglaunchWidth + launchIndex.x) * max_photon_count;
    float sample[6];
    haltonSample(pm_index, sample);
    //rtPrintf("3");
    optix::Ray photonRay;
    float3 N1;
    float pdf;
    CudaSpectrum Le=Sample_L(lightSourceIndex, sample[LU1], sample[LU2], 
        sample[U1], sample[U2], &photonRay, &N1, &pdf);
    if (pdf==0.0f || isBlack(Le)) return;
    for (uint i=0; i<max_photon_count; ++i){
        setInValid(indirectPhotonmap[pm_index+i]);
    }
    photonRay.ray_type=PM_PhotonTracingType;
    photonRay.tmax=RT_DEFAULT_MAX;
    CudaSpectrum alpha = (AbsDot(N1, photonRay.direction) * Le) / (pdf*photonTracingEmittingPhotons);
    if(isPrint()){
        rtPrintf("\nclpha: %f %f %f, Le: %f %f %f, dot: %f pdf:%f", 
            alpha.x, alpha.y, alpha.z, Le.x, Le.y, Le.z, AbsDot(N1, photonRay.direction), pdf);
    }
    PhotonTraingPayLoad pld={alpha, true, 0, pm_index};
#ifdef DEBUG_KERNEL
    if (isPrint()){
        debugPrint(photonRay);
    }
#endif
    rtTrace(top_group, photonRay, pld);
}
__inline__ __device__ bool materialHasNonSpecular()
{
    return true;
}

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(PhotonTraingPayLoad, photonTracingPayLoad, rtPayload, );

RT_PROGRAM void photontracing_closest_hit()
{
    float3 hit_point = ray.origin + t*ray.direction;
    float3 world_shading_normal   = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));

    if (isSpecular(materialType)){
        Ray newRay;
        CudaSpectrum spec=materialSpecular(-ray.direction, &newRay.direction, false,hit_point);
        newRay.origin=hit_point;
        newRay.ray_type=PM_PhotonTracingType;
        newRay.tmin=scene_epsilon;
        newRay.tmax=RT_DEFAULT_MAX;
        photonTracingPayLoad.alpha*=spec;
        if(photonTracingPayLoad.nIntersections==0){
            photonTracingPayLoad.nIntersections++;
        }
        rtTrace(top_group, newRay, photonTracingPayLoad);
        return;
    }

    float3 world_geometric_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, geometry_normal));
    float3 wo = -ray.direction;
    //float3 ffnormal = faceforward(world_shading_normal, wo, world_geometric_normal);
#ifdef DEBUG_KERNEL
    if(isPrint()){
        rtPrintf("\n\nwhadingNormal:%f %f %f", world_shading_normal.x, world_shading_normal.y, world_shading_normal.z);
    }
#endif
    
    uint nIntersections=photonTracingPayLoad.nIntersections;
    if (materialHasNonSpecular()){
        //bool depositedPhoton = false;
        if (nIntersections >= 1) { //at least bounced once
            CudaPhoton& photon=indirectPhotonmap[photonTracingPayLoad.pm_index+nIntersections-1];
            setValid(photon);
            photon.p=hit_point;
            photon.alpha= photonTracingPayLoad.alpha;
            photon.wi=wo;
            //depositedPhoton = true;
        }
    }
    
    if (nIntersections >= max_photon_count) {
#ifdef DEBUG_KERNEL
        if(isPrint())
            rtPrintf("\nr");
#endif
        return;
    }

    float3 wi;
    float pdf;
    uint ranomIndex=3*(photonTracingPayLoad.pm_index+nIntersections);
    float u1=photonTracingRandom[ranomIndex];
    float u2=photonTracingRandom[ranomIndex+1];
    CudaSpectrum fr = Sample_f(wo, &wi, u1, u2, &pdf);

    if (isBlack(fr) || pdf == 0.f) return;
    CudaSpectrum anew = photonTracingPayLoad.alpha * fr *
        AbsDot(wi, shading_normal) / pdf;

#ifdef DEBUG_KERNEL
    if(isPrint())
        rtPrintf("\nanew:%f %f %f", anew.x, anew.y, anew.z);
#endif

    // Possibly terminate photon path with Russian roulette
    /*float continueProb = min(1.f, anew.y / photonTracingPayLoad.alpha.y); //the pbrt use the y() value
    if (photonTracingRandom[ranomIndex+2] > continueProb)
        return;
    photonTracingPayLoad.alpha = anew / continueProb;
    */
    photonTracingPayLoad.alpha=anew;
    photonTracingPayLoad.nIntersections++;

    Ray newRay = Ray(hit_point, wi, PM_PhotonTracingType,
        scene_epsilon);
#ifdef DEBUG_KERNEL
    if(isPrint()){
        debugPrint(newRay);
    }
#endif 
    rtTrace(top_group, newRay, photonTracingPayLoad);
}

RT_PROGRAM void  photontracing_exception()
{
    rtPrintExceptionDetails();
}

RT_PROGRAM void  photontracing_miss()
{
#ifdef DEBUG_KERNEL
    if(isPrint()){
        rtPrintf("\nm");
    }
#endif
}

RT_PROGRAM void  photontracing_debug_anyhit()
{
#ifdef DEBUG_KERNEL
    if(isPrint()){
        rtPrintf("\na");
    }
#endif
}