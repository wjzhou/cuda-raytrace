#include "hip/hip_runtime.h"
#include "util/camera/camera.cu.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "util/light/cudalight.cu.h"
#include "util/camera/camera.cu.h"
#include "util/material/cudamaterial.cu.h"
#include "photonmapping.h"
using namespace optix;

struct RtRayPayload
{
    optix::uint ray_depth;
};

rtDeclareVariable(rtObject, top_group, ,);
rtDeclareVariable(float, scene_epsilon, ,);
rtBuffer<CudaRayDifferential, 2> bRays;

RT_PROGRAM void raytracing_camera(){
    CudaRayDifferential rd=bRays[launchIndex];
    optix::Ray ray(rd.o, rd.d, PM_RayTracingType, scene_epsilon);
    RtRayPayload pld;
    pld.ray_depth=0u;
    rtTrace(top_group, ray, pld);
}

__device__ __inline__ float3 WorldtoLocal(const float3& v,
                                          const float3& nn, const float3& sn, const float3& tn)
{
    return make_float3(dot(v, sn), dot(v,tn), dot(v,nn));
}

struct ShadowPRD
{
    float attenuation;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RtRayPayload, raytracingPayLoad, rtPayload, );
rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );
//rtDeclareVariable(int, lightsource,,); //if the shape is not a light source
// this is  -1, otherwise the index to light source

//rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);
//rtDeclareVariable(MaterialType, materialType, ,);

rtDeclareVariable(int, lightIndex, ,);
__device__ __inline__ CudaSpectrum directLight(const RayTracingRecord& rec){
    CudaSpectrum L=black();
    const float3& point=rec.position;

    float3 world_shading_normal = rec.shadingNormal;
    //    float3 world_geometry_normal = rec.geometryNormal;


    int totalLight=lightSize();

    if(lightIndex>=totalLight){
        rtPrintf("error, out of bound light index:%d", lightIndex);
        return L; //prevent GPU kernel crash
    }

    L+=lightL(lightIndex, -rec.direction);

    for (int i=0; i<totalLight; ++i)
    {
        int nSamples=lightNSamples(i);
        for (int iSample=0; iSample<nSamples; ++iSample){
            float3 uwi;
            float pdf;
            CudaSpectrum li=Sample_L(i, point, uwi, pdf, iSample);
            Ray shadowRay(point, uwi, PM_ShadowRayType, 0.001f, 1.0f-0.001f);
            ShadowPRD pld;
            pld.attenuation=1.0f;
            rtTrace(top_group, shadowRay, pld);
            float3 wi=normalize(uwi);
            float3 wo=normalize(-rec.direction);
            L+=pld.attenuation*fabs(dot(world_shading_normal, wi))*f(rec.material, rec.materialParameter, wo, wi)*li/(pdf*nSamples);
            //rtPrintf("attenuation:%f dot:%f lix:%f pdf:%f", pld.attenuation, fabs(dot(world_shading_normal, wi)), li.x, pdf);
        }
    }
    return L;
}

rtBuffer<RayTracingRecord, 2> bRayTracingOutput;
RT_PROGRAM void raytracing_closest_hit()
{
    const float3 point=ray.origin+ray.direction*t;
    if (isSpecular(materialType)){
        Ray newRay;
        CudaSpectrum spec=materialSpecular(-ray.direction, &newRay.direction, true, point);
        newRay.origin=point;
        newRay.ray_type=PM_RayTracingType;
        newRay.tmin=scene_epsilon;
        newRay.tmax=RT_DEFAULT_MAX;
        raytracingPayLoad.ray_depth++;
        if (raytracingPayLoad.ray_depth>10){
            bRayTracingOutput[launchIndex].flags=RayTracingRecordFlageException;
            return;
        }
        rtTrace(top_group, newRay, raytracingPayLoad);
        return;
    }

    RayTracingRecord& record=bRayTracingOutput[launchIndex];

    record.flags=0;
    record.position=point;
    record.dpdu=normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, aDpdu));
    record.dpdv=normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, aDpdv));
    record.shadingNormal=normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, aShadingNormal));
    record.geometryNormal=normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, aGeometryNormal));
    record.direction=ray.direction;
    record.material=materialType;
    record.materialParameter=materialParameter;
    record.flux=black();
    record.photon_count=0;
    record.radius2=4.f;
    CudaSpectrum DL=directLight(record);
    //CudaSpectrum DL=black();
    rtPrintf("\nDL: %f, %f, %f", DL.x, DL.y, DL.z);
    record.directLight=DL;
}

RT_PROGRAM void raytracing_miss()
{
    bRayTracingOutput[launchIndex].flags=RayTracingRecordFlageMISS;
}


RT_PROGRAM void raytracing_exception()
{
    bRayTracingOutput[launchIndex].flags=RayTracingRecordFlageException;
    rtPrintExceptionDetails();
}


RT_PROGRAM void shadow_any_hit()
{
    shadow_prd.attenuation=0.f;
    rtTerminateRay();
}

