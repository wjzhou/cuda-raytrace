#include "hip/hip_runtime.h"
#include "util/camera/camera.cu.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "util/light/cudalight.cu.h"
#include "util/camera/camera.cu.h"
#include "util/material/cudamaterial.cu.h"
#include "photonmapping.h"
using namespace optix;

struct RtRayPayload
{
     optix::uint ray_depth;
};

rtDeclareVariable(rtObject, top_group, ,);
rtDeclareVariable(float, scene_epsilon, ,);
rtBuffer<CudaRayDifferential, 2> bRays;

RT_PROGRAM void raytracing_camera(){
    CudaRayDifferential rd=bRays[launchIndex];
    optix::Ray ray(rd.o, rd.d, PM_RayTracingType, scene_epsilon);
    RtRayPayload pld;
    pld.ray_depth=0u;
    rtTrace(top_group, ray, pld);
}

__device__ __inline__ float3 WorldtoLocal(const float3& v,
    const float3& nn, const float3& sn, const float3& tn)
{
    return make_float3(dot(v, sn), dot(v,tn), dot(v,nn));
}

struct ShadowPRD
{
  float attenuation;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t, rtIntersectionDistance, );
rtDeclareVariable(RtRayPayload, raytracingPayLoad, rtPayload, );
rtDeclareVariable(ShadowPRD, shadow_prd, rtPayload, );
//rtDeclareVariable(int, lightsource,,); //if the shape is not a light source
                                      // this is  -1, otherwise the index to light source

//rtDeclareVariable(uint2, launchIndex, rtLaunchIndex,);
//rtDeclareVariable(MaterialType, materialType, ,);
rtBuffer<RayTracingRecord, 2> bRayTracingOutput;
RT_PROGRAM void raytracing_closest_hit()
{
    const float3 point=ray.origin+ray.direction*t;
    if (isSpecular(materialType)){
        Ray newRay;
        CudaSpectrum spec=materialSpecular(-ray.direction, &newRay.direction, true, point);
        newRay.origin=point;
        newRay.ray_type=PM_RayTracingType;
        newRay.tmin=scene_epsilon;
        newRay.tmax=RT_DEFAULT_MAX;
        raytracingPayLoad.ray_depth++;
        if (raytracingPayLoad.ray_depth>10){
            bRayTracingOutput[launchIndex].flags=RayTracingRecordFlageException;
            return;
        }
        rtTrace(top_group, newRay, raytracingPayLoad);
        return;
    }

    RayTracingRecord record;
    float3 world_shading_normal = normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometry_normal = normalize(
        rtTransformNormal(RT_OBJECT_TO_WORLD, geometry_normal));
    
    record.flags=0;
    record.position=point;
    record.dpdu=dpdu;
    record.dpdv=dpdv;
    record.shadingNormal=shading_normal;
    record.geometryNormal=geometry_normal;
    record.direction=ray.direction;
    record.material=materialType;
    record.materialParameter=materialParameter;

    bRayTracingOutput[launchIndex]=record;
}

RT_PROGRAM void raytracing_miss()
{
//bOutput[launchIndex]=make_float3(0.3f, 0.3f, 0.0f);
    bRayTracingOutput[launchIndex].flags=RayTracingRecordFlageMISS;
}


RT_PROGRAM void raytracing_exception()
{
  bRayTracingOutput[launchIndex].flags=RayTracingRecordFlageException;
  rtPrintExceptionDetails();
}


RT_PROGRAM void shadow_any_hit()
{
    shadow_prd.attenuation=0.f;
    rtTerminateRay();
}

